#include "hip/hip_runtime.h"
#include "matrices.cuh"
#include <stdlib.h>
#include <assert.h>

#define checkCUDAError(val) { checkError((val), #val, __FILE__, __LINE__); }    // in-line regular function

void checkError(hipError_t code, char const * func, const char *file, const int line)
{
    if (code != hipSuccess) 
    {
        std::cerr << "CUDA error returned from \"" << func << "\" at "
                  << file << ":" << line << "\nError code: " << code
                  << "(" << hipGetErrorString(code) << ")\n";
        hipDeviceReset();
        exit(code);
    }
}

__global__ void mat_init(float* buffer, int height, int width, int value) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i >= width * height) return;

    buffer[i] = value;
}

Mat::Mat(int height, int width)
    : m_height(height)
    , m_width(width)
    , m_buffer((float*) calloc(height * width, sizeof(float)))
{}

// Need to use a custom kernel instead of CudaMemSet because we operate of float pointers
Mat::Mat(int height, int width, float value)
    : m_height{height}
    , m_width{width}
{
    std::size_t buffer_size = height * width;
    this->m_buffer = (float*) malloc(height * width * sizeof(float));
    float* d_buffer;
    checkCUDAError(hipMalloc(&d_buffer, height * width * sizeof(float)));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock) ? buffer_size : prop.maxThreadsPerBlock;
    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;
    mat_init<<<nbBlocks, threadsPerBlock>>>(d_buffer, height, width, value);
    hipDeviceSynchronize();
    checkCUDAError(hipMemcpy(this->m_buffer, d_buffer, height * width * sizeof(float), hipMemcpyDeviceToHost));
    hipFree(d_buffer);
}

Mat::Mat(float* list_init, int height, int width)
    : m_height(height)
    , m_width(width)
{
    std::size_t buffer_size = height * width;
    this->m_buffer = (float*) malloc(buffer_size * sizeof(float));
    checkCUDAError(hipMemcpy(this->m_buffer, list_init, buffer_size * sizeof(float), hipMemcpyHostToHost));
}

Mat::Mat(float* list_init, int height)
    : Mat(list_init, height, 1)
    {}

Mat::Mat(const Mat& m)
    : Mat(m.m_buffer, m.m_height, m.m_width)
    {}

Mat Mat::copy() const
{ return Mat(m_buffer, m_height, m_width);}


void Mat::operator=(const Mat& other)
{
    m_height = other.m_height;
    m_width = other.m_width;
    checkCUDAError(hipMemcpy(m_buffer, other.m_buffer,
                              m_height * m_width * sizeof(float), hipMemcpyHostToHost));
}

Mat::~Mat(){
    free(this->m_buffer);
}

// I don't think that using a kernel (with all the overhead needed) will be faster than a little for loop
Mat Mat::eye(int dim)
{
    Mat ret(dim, dim);
    for (int i = 0; i < dim; ++i)
        ret.m_buffer[i * ret.m_width + i] = 1;
    return ret;
}

// Internet say, use a loop for k to avoid concurrency problem
__global__ void dot_kernel(float* self, float* other, float* ret,
                           int s_height, int s_width, int o_width){
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * o_width) return;

    int i = th / o_width; //0 to height
    int j = th % o_width; //0 to width
    for (int k = 0; k < s_width; ++k)
        ret[i * o_width + j] += self[i * s_width + k] * other[k * o_width + j];
}

Mat Mat::dot(const Mat& other)
{
    if (m_width != other.m_height)
    {
        printf("Invalid dot product, shapes do not match {%i, %i} vs {%i, %i}",
               m_height, m_width, other.m_height, other.m_width);
        throw "Invalid dot product";
    }

    Mat ret(m_height, other.m_width);
    float* ret_buffer;
    checkCUDAError(hipMalloc(&ret_buffer, ret.m_height * ret.m_width* sizeof(float)));

    float* self_buffer;
    checkCUDAError(hipMalloc(&self_buffer, m_height * m_width* sizeof(float)));
    checkCUDAError(hipMemcpy(self_buffer, m_buffer, m_height * m_width * sizeof(float), hipMemcpyHostToDevice));
    float* other_buffer;
    checkCUDAError(hipMalloc(&other_buffer, other.m_height * other.m_width * sizeof(float)));
    checkCUDAError(hipMemcpy(other_buffer, other.m_buffer,
                              other.m_height * other.m_width * sizeof(float), hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t buffer_size = ret.m_height * ret.m_width;
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock)
        ? buffer_size : prop.maxThreadsPerBlock;
    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;
    dot_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, other_buffer, ret_buffer,
                                              m_height, m_width, other.m_width);
    hipDeviceSynchronize();

    checkCUDAError(hipMemcpy(ret.m_buffer, ret_buffer, ret.m_height * ret.m_width * sizeof(float),
                              hipMemcpyDeviceToHost));

    hipFree(ret_buffer);
    hipFree(self_buffer);
    hipFree(other_buffer);
    return ret;
}

__global__ void T_kernel(float* self, float* ret, int s_height, int s_width) {
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * s_width) return;

    int i = th / s_width; //0 to height
    int j = th % s_width; //0 to width
    ret[j * s_height + i] = self[i * s_width + j];
}

Mat Mat::T() {
    Mat ret(m_width, m_height);
    float* ret_buffer;
    checkCUDAError(hipMalloc(&ret_buffer, ret.m_height * ret.m_width * sizeof(float)));

    float* self_buffer;
    checkCUDAError(hipMalloc(&self_buffer, m_height * m_width* sizeof(float)));
    checkCUDAError(hipMemcpy(self_buffer, m_buffer, m_height * m_width * sizeof(float), hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t buffer_size = ret.m_height * ret.m_width;
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock)
        ? buffer_size : prop.maxThreadsPerBlock;
    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;
    T_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, ret_buffer, m_height, m_width);
    hipDeviceSynchronize();

    checkCUDAError(hipMemcpy(ret.m_buffer, ret_buffer, ret.m_height * ret.m_width * sizeof(float),
                              hipMemcpyDeviceToHost));

    hipFree(ret_buffer);
    hipFree(self_buffer);
    return ret;
}

__global__ void add_kernel(float* self, float* other, float* ret, int s_height, int s_width) {
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * s_width) return;
    ret[th] = self[th] + other[th];
}

__global__ void add_broadcast_kernel(float* self, float* other, float* ret, int s_height, int s_width) {
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * s_width) return;
    int i = th / s_width; //0 to height
    int j = th % s_width; //0 to width
    ret[i * s_width + j] = self[i * s_width + j] + other[j];
}

Mat Mat::operator+(const Mat& other) const{
    if ((this->m_width != other.m_width) || (m_height != other.m_height && other.m_height != 1))
    {
        printf("Could not add matrices, dimensions do not match {%i, %i} vs {%i, %i}",
            this->m_height, this->m_width, other.m_height, other.m_width);
        throw "Invalid addition";
    }


    Mat ret(m_height, m_width);
    float* ret_buffer;
    checkCUDAError(hipMalloc(&ret_buffer, ret.m_height * ret.m_width* sizeof(float)));

    float* self_buffer;
    checkCUDAError(hipMalloc(&self_buffer, m_height * m_width* sizeof(float)));
    checkCUDAError(hipMemcpy(self_buffer, m_buffer, m_height * m_width * sizeof(float), hipMemcpyHostToDevice));
    float* other_buffer;
    checkCUDAError(hipMalloc(&other_buffer, other.m_height * other.m_width * sizeof(float)));
    checkCUDAError(hipMemcpy(other_buffer, other.m_buffer,
                              other.m_height * other.m_width * sizeof(float), hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t buffer_size = ret.m_height * ret.m_width;
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock)
        ? buffer_size : prop.maxThreadsPerBlock;
    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;

    if (m_height == other.m_height)
        add_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, other_buffer, ret_buffer,
                                                  m_height, m_width);
    else
        add_broadcast_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, other_buffer, ret_buffer,
                                                  m_height, m_width);

    hipDeviceSynchronize();

    checkCUDAError(hipMemcpy(ret.m_buffer, ret_buffer, ret.m_height * ret.m_width * sizeof(float),
                              hipMemcpyDeviceToHost));

    hipFree(ret_buffer);
    hipFree(self_buffer);
    hipFree(other_buffer);
    return ret;
}

__global__ void sub_kernel(float* self, float* other, float* ret, int s_height, int s_width) {
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * s_width) return;
    ret[th] = self[th] - other[th];
}

__global__ void sub_broadcast_kernel(float* self, float* other, float* ret, int s_height, int s_width) {
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * s_width) return;
    int i = th / s_width; //0 to height
    int j = th % s_width; //0 to width
    ret[i * s_width + j] = self[i * s_width + j] - other[j];
}

Mat Mat::operator-(const Mat& other) const{
    if ((this->m_width != other.m_width) || (m_height != other.m_height && other.m_height != 1))
    {
        printf("Could not add matrices, dimensions do not match {%i, %i} vs {%i, %i}",
            this->m_height, this->m_width, other.m_height, other.m_width);
        throw "Invalid addition";
    }


    Mat ret(m_height, m_width);
    float* ret_buffer;
    checkCUDAError(hipMalloc(&ret_buffer, ret.m_height * ret.m_width* sizeof(float)));

    float* self_buffer;
    checkCUDAError(hipMalloc(&self_buffer, m_height * m_width* sizeof(float)));
    checkCUDAError(hipMemcpy(self_buffer, m_buffer, m_height * m_width * sizeof(float), hipMemcpyHostToDevice));
    float* other_buffer;
    checkCUDAError(hipMalloc(&other_buffer, other.m_height * other.m_width * sizeof(float)));
    checkCUDAError(hipMemcpy(other_buffer, other.m_buffer,
                              other.m_height * other.m_width * sizeof(float), hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t buffer_size = ret.m_height * ret.m_width;
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock)
        ? buffer_size : prop.maxThreadsPerBlock;
    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;

    if (m_height == other.m_height)
        sub_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, other_buffer, ret_buffer,
                                                  m_height, m_width);
    else
        sub_broadcast_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, other_buffer, ret_buffer,
                                                  m_height, m_width);

    hipDeviceSynchronize();

    checkCUDAError(hipMemcpy(ret.m_buffer, ret_buffer, ret.m_height * ret.m_width * sizeof(float),
                              hipMemcpyDeviceToHost));

    hipFree(ret_buffer);
    hipFree(self_buffer);
    hipFree(other_buffer);
    return ret;
}

__global__ void normalize_kernel(float *A, float *I, int n, int x, bool diag){
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= n * n) return;
    int i = th / n; //0 to height
    int j = th % n; //0 to width
    if ((!diag && (i == x && i != j)) || (diag && (i == x && i == j))){
        I[i * n + j] /= A[x * n + x];
        A[i * n + j] /= A[x * n + x];
    }
}

__global__ void gaussjordan_kernel(float *A, float *I, int n, int x)
{
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= n * n) return;
    int i = th / n; //0 to height
    int j = th % n; //0 to width

    if (i != x) {
        I[i * n + j] -= I[x * n + j] * A[i * n + x];
        if (j != x){
            A[i * n + j] -= A[x * n + j] * A[i * n + x];
        }
    }
}

__global__ void zero_kernel(float *A, int n, int x){
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= n * n) return;
    int i = th / n; //0 to height
    int j = th % n; //0 to width

    if (i != x && j == x){
        A[i * n + j] = 0;
    }
}

Mat Mat::inverse() const
{
    Mat ret = eye(m_height);
    float* ret_buffer;
    checkCUDAError(hipMalloc(&ret_buffer, ret.m_height * ret.m_width * sizeof(float)));
    checkCUDAError(hipMemcpy(ret_buffer, ret.m_buffer, ret.m_height * ret.m_width * sizeof(float),
                              hipMemcpyHostToDevice));

    float* self_buffer;
    checkCUDAError(hipMalloc(&self_buffer, m_height * m_width * sizeof(float)));
    checkCUDAError(hipMemcpy(self_buffer, m_buffer, m_height * m_width * sizeof(float), hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t buffer_size = ret.m_height * ret.m_width;
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock)
        ? buffer_size : prop.maxThreadsPerBlock;
    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;

    for (int i = 0; i < m_height; ++i)
    {
        normalize_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, ret_buffer,
                                                        m_height, i, 0);
        normalize_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, ret_buffer,
                                                        m_height, i, 1);
        gaussjordan_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, ret_buffer,
                                                        m_height, i);
        zero_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, m_height, i);
    }

    checkCUDAError(hipMemcpy(ret.m_buffer, ret_buffer, ret.m_height * ret.m_width * sizeof(float),
                              hipMemcpyDeviceToHost));

    hipFree(self_buffer);
    hipFree(ret_buffer);

    return ret;
}

void Mat::print() const {
    std::cout << "{\n";
    for (int i = 0; i < this->m_height; ++i) {
        std::cout << "  { ";
        for (int j = 0; j < this->m_width;) {
            std::cout << this->m_buffer[i * this->m_width + j];
            if (++j < this->m_width)
                std::cout << ", ";
        }
        std::cout << " }\n";
    }
    std::cout << "}\n";
}