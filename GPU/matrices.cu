#include "hip/hip_runtime.h"
#include "matrices.cuh"
#include <assert.h>

__global__ void mat_init(float** buffer, int height, int width, int value) {
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    int j = blockDim.y*blockIdx.y + threadIdx.y;
    if (i >= height || j >= width) return;

    (*buffer)[i * height + j] = value;
}

Mat::Mat(int height, int width)
    : m_height{height}
    , m_width{width}
{
    this->m_buffer = (float*)malloc(height * width * sizeof(float));
    float* d_buffer = NULL;
    hipMalloc((void **)&d_buffer, height * width * sizeof(float));
    
    mat_init<<<1, 6>>>(&d_buffer, height, width, 0);
    hipMemcpy(this->m_buffer, d_buffer, height*width*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_buffer);
}

Mat::Mat(int height, int width, float value)
    : m_height{height}
    , m_width{width}
{
    this->m_buffer = (float*)malloc(height * width * sizeof(float));
    float* d_buffer = NULL;
    hipMalloc((void **)&d_buffer, height * width * sizeof(float));

    mat_init<<<1, 6>>>(&d_buffer, height, width, value);
    hipMemcpy(this->m_buffer, d_buffer, height*width*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_buffer);
}

// __global__ Mat::Mat(const float** list_init){
//     auto height = list_init.size();
//     auto width = list_init[0].size();

//     for (std::size_t i = 0; i < height; ++i)
//         if (list_init[i].size() != width)
//             throw "Invalid list initialization, internal vectors were not of same width.";

//     m_height = height;
//     m_width = width;
//     m_buffer = list_init;
// }

// __global__ Mat::Mat(const float* list_init, int height){ // Need to fix
//     this->m_width = 1;
//     this->m_height = height;
//     for (int i = 0; i < height; ++i)
//         m_buffer.push_back(std::vector<float>{list_init[i]});
// }

// __global__ Mat::Mat(const Mat& m)
//     : m_height(m.m_height)
//     , m_width(m.m_width)
// {
//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;
//     if (i >= m.m_height || j >= m.m_width) return;
 
//     if (i == 0 && j == 0) {
//         auto err = hipMalloc(&this->m_buffer, m.m_height*m.m_width*sizeof(float));
//         assert(err != NULL);
//     }
//     this->m_buffer[i][j] = m.m_buffer[i][j];
// }

// Mat Mat::eye(int dim)
// {
//     Mat ret(dim, dim);
//     for (int i = 0; i < dim; ++i)
//         ret[i][i] = 1;
//     return ret;
// }

// __global__ void Mat::dot(const Mat& other, Mat* ret){
//     if (this->m_width != other.m_height)
//     {
//         printf("Invalid dot product, shapes do not match {%i, %i} vs {%i, %i}",
//             this->m_height, this->m_width, other.m_height, other.m_width);
//         assert(this->m_width == other.m_height);
//     }
//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;
//     int k = blockDim.z*blockId.z + threadId.z;

//     if (i == 0 && j == 0 && k == 0) {
//         // Initialize the return matrix
//         ret = &Mat(this->m_height, other.m_width);
//     }

//     if (i >= this->m_height || j >= this->m_width || k >= this->m_height) return;
//     ret->m_buffer[i][j] += this->m_buffer[i][k] * other.m_buffer[k][j];
// }

// void Mat::dot(const float* other, int height, Mat* ret)
// {
//     if ((std::size_t)this->m_width != other.size())
//     {
//         printf("Invalid dot product, shapes do not match {%i, %i} vs {%zd, 1}",
//                this->m_height, this->m_width, other.size());
//         throw "Invalid dot product";
//     }
//     Mat vector(other, height);
//     dot(vector, ret); // Fix dot call
// }

// __global__ void Mat::T(Mat* ret) {
//     if (ret == NULL) {
//         // Initialize the return matrix
//         ret = &Mat(this->m_width, this->m_height);
//     }

//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;
//     if (i >= this->m_height || j >= this->m_width) return;
//     ret->m_buffer[j][i] = this->m_buffer[i][j];
// }

// float* Mat::operator[](const int pos) const {
//     return this->m_buffer[pos];
// }

// const float* Mat::operator[](const int pos) {
//     return this->m_buffer[pos];
// }

// __global__ void Mat::operator+(const Mat& other, Mat* ret) const{
//     if ((this->m_width != other.m_width) || (m_height != other.m_height && other.m_height != 1))
//     {
//         printf("Could not add matrices, dimensions do not match {%i, %i} vs {%i, %i}",
//             this->m_height, this->m_width, other.m_height, other.m_width);
//         throw "Invalid addition";
//     }

//     if (ret == NULL) {
//         // Initialize the return matrix
//         ret = Mat(this->m_height, this->m_width);
//     }

//     if (m_height == other.m_height)
//     {
//         int i = blockDim.x*blockId.x + threadId.x;
//         int j = blockDim.y*blockId.y + threadId.y;
//         if (i >= this->m_height || j >= this->m_width) return;    
//         ret->m_buffer[i][j] = this->m_buffer[i][j] + other.m_buffer[i][j];
//     }
//     else
//     {
//         int i = blockDim.x*blockId.x + threadId.x;
//         int j = blockDim.y*blockId.y + threadId.y;
//         if (i >= this->m_height || j >= this->m_width) return;    
//         ret->m_buffer[i][j] = this->m_buffer[i][j] + other.m_buffer[0][j];
//     }
// }

// __global__ void Mat::operator-(const Mat& other, Mat* ret) const{
//     if ((this->m_width != other.m_width) || (m_height != other.m_height && other.m_height != 1))
//     {
//         printf("Could not subtract matrices, dimensions do not match {%i, %i} vs {%i, %i}",
//             this->m_height, this->m_width, other.m_height, other.m_width);
//         throw "Invalid subtraction";
//     }

//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;
    
//     if (i == 0 && j == 0) {
//         // Initialize the return matrix
//         ret = Mat(this->m_height, this->m_width);
//     } 
//     else if (i >= this->m_height || j >= this->m_width) return;
    
//     if (m_height == other.m_height) {
//         ret->m_buffer[i][j] = this->m_buffer[i][j] - other.m_buffer[i][j];
//     }
//     else {
//         ret->m_buffer[i][j] = this->m_buffer[i][j] - other.m_buffer[0][j];
//     }
// }

// __global__ void Mat::operator*(const float& factor, Mat* ret) const{
//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;

//     if (i == 0 && j == 0) {
//         // Initialize the return matrix
//         ret = Mat(this->m_height, this->m_width);
//     } 
//     else if (i >= this->m_height || j >= this->m_width) return;

//     ret.m_buffer[i][j] = factor * this->m_buffer[i][j];
// }

// __global__ void Mat::operator*(const Mat& other, Mat* ret) const{
//     /**
//      * Trying to replicate numpy broadcasting 
//      * https://numpy.org/doc/stable/user/basics.broadcasting.html
//      * TODO
//      */
//     int ret_h = this->m_height;
//     if (other.m_height > ret_h) {
//         ret_h = other.m_height;
//     }
//     int ret_w = this->m_height;
//     if (other.m_height > ret_w) {
//         ret_w = other.m_width;
//     }
    
//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;

//     if (i == 0 && j == 0) {
//         // Initialize the return matrix
//         ret = Mat(ret_h, ret_w);
//     } else if (i >= ret_h || j >= ret_w) return;


//     if (this->m_height == 1 && other.m_width == 1){
//         ret->m_buffer[i][j] += this->m_buffer[0][i] * other.m_buffer[j][0];
//     }
//     else if (this->m_width == 1 && other.m_height == 1){
//         ret.m_buffer[i][j] += other.m_buffer[0][i] * this->m_buffer[j][0];
//     }
//     else{
//         printf("Could not broadcast matrices, dimensions do not match {%i, %i} vs {%i, %i}",
//                this->m_height, this->m_width, other.m_height, other.m_width);
//         assert(0); // Need cleaner asset
//     }
// }

void Mat::print() const {
    std::cout << "{\n";
    for (int i = 0; i < this->m_height; ++i) {
        std::cout << "  { ";
        for (int j = 0; j < this->m_width;) {
            std::cout << this->m_buffer[i][j];
            if (++j < this->m_width)
                std::cout << ", ";
        }
        std::cout << " }\n";
    }
    std::cout << "}\n";
}

// __global__ void Mat::copy(Mat* ret) const {
//     if (ret == NULL) {
//         // Initialize the return matrix
//         ret = Mat(this->m_height, this->m_width);
//     }

//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;
//     if (i >= this->m_height || j >= this->m_width) return;    
//     ret->m_buffer[h][w] = this->m_buffer[h][w];
// }

// float mean_vector(const std::vector<float>& v)
// {
//     float r = 0.;
//     for (std::size_t i = 0; i < v.size(); ++i)
//         r += v[i];
//     return r / v.size();
// }

// Mat Mat::mean() const {
//     std::vector<float> aggreagate(m_width, 0);
//     for (int i = 0; i < m_height; ++i)
//         for (int j = 0; j < m_width; ++j)
//             aggreagate[j] += m_buffer[i][j];
//     for (int i = 0; i < m_width; ++i)
//         aggreagate[i] /= m_height;
//     return Mat(std::vector<std::vector<float>>{aggreagate});
// }

// std::vector<std::tuple<float, Eigen::VectorXf>> get_eigen(Eigen::MatrixXf m) {
//     Eigen::EigenSolver<Eigen::MatrixXf> eigensolver;

//     eigensolver.compute(m);

//     Eigen::VectorXf eigen_values = eigensolver.eigenvalues().real();
//     Eigen::MatrixXf eigen_vectors = eigensolver.eigenvectors().real();
//     std::vector<std::tuple<float, Eigen::VectorXf>> eigen_vectors_and_values;

//     for(int i = 0; i < eigen_values.size(); i++){
//         std::tuple<float, Eigen::VectorXf> vec_and_val(eigen_values[i], eigen_vectors.row(i));
//         eigen_vectors_and_values.push_back(vec_and_val);
//     }

//     std::sort(eigen_vectors_and_values.begin(), eigen_vectors_and_values.end(),
//               [&](const std::tuple<float, Eigen::VectorXf>& a, const std::tuple<float, Eigen::VectorXf>& b) -> bool{
//                   return std::get<0>(a) <= std::get<0>(b);
//               });

//     return eigen_vectors_and_values;
// }

// std::vector<std::tuple<float, std::vector<float>>> Mat::eigen() const {
//     Eigen::MatrixXf eigen_mat(m_height, m_width);

//     for (int i = 0; i < m_height; ++i)
//         for (int j = 0; j < m_width; ++j)
//             eigen_mat(i, j) = m_buffer[i][j];

//     auto eigen_value_vector = get_eigen(eigen_mat);

//     std::vector<std::tuple<float, std::vector<float>>> ret;
//     for (std::size_t i = 0; i < eigen_value_vector.size(); ++i)
//     {
//         std::vector<float> tmp;
//         auto eigen_vector = std::get<1>(eigen_value_vector[i]);
//         for (int j = 0; j < m_height; ++j)
//             tmp.push_back(eigen_vector[j]);
//         std::tuple<float, std::vector<float>> tup = std::make_tuple(std::get<0>(eigen_value_vector[i]), tmp);
//         ret.push_back(tup);
//     }

//     return ret;
// }

// Mat Mat::inverse() const {
//     Eigen::MatrixXf eigen_mat(m_height, m_width);

//     for (int i = 0; i < m_height; ++i)
//         for (int j = 0; j < m_width; ++j)
//             eigen_mat(i, j) = m_buffer[i][j];


//     auto eigen_inverse = eigen_mat.inverse();

//     Mat ret(m_height, m_width);
//     for (int i = 0; i < m_height; ++i) {
//         for (int j = 0; j < m_width; ++j) {
//             ret[i][j] = eigen_inverse(i, j);
//         }
//     }

//     return ret;
// }
