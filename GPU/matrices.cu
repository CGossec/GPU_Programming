#include "hip/hip_runtime.h"
#include "matrices.cuh"
#include <assert.h>

__global__ void mat_init(float* buffer, int height, int width, int value) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i >= width * height) return;

    buffer[i] = value;
}

Mat::Mat(int height, int width)
    : m_height(height)
    , m_width(width)
    , m_buffer((float*) calloc(height * width, sizeof(float)))
{}

// Need to use a custom kernel instead of CudaMemSet because we operate of float pointers
Mat::Mat(int height, int width, float value)
    : m_height{height}
    , m_width{width}
{
    std::size_t buffer_size = height * width;
    this->m_buffer = (float*) malloc(height * width * sizeof(float));
    float* d_buffer = NULL;
    hipMalloc((void **)&d_buffer, height * width * sizeof(float));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock) ? buffer_size : prop.maxThreadsPerBlock;

    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;
    mat_init<<<nbBlocks, threadsPerBlock>>>(d_buffer, height, width, value);
    hipDeviceSynchronize();
    hipMemcpy(this->m_buffer, d_buffer, height * width * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_buffer);
}

Mat::Mat(float* list_init, int height, int width)
    : m_height(height)
    , m_width(width)
{
    std::size_t buffer_size = height * width;
    this->m_buffer = (float*) malloc(buffer_size * sizeof(float));
    hipMemcpy(this->m_buffer, list_init, buffer_size * sizeof(float), hipMemcpyHostToHost);
}

Mat::Mat(float* list_init, int width)
    : Mat(list_init, 1, width)
    {}

Mat::Mat(const Mat& m)
    : Mat(m.m_buffer, m.m_height, m.m_width)
    {}

Mat::~Mat(){
    free(this->m_buffer);
}

// Mat Mat::eye(int dim)
// {
//     Mat ret(dim, dim);
//     for (int i = 0; i < dim; ++i)
//         ret[i][i] = 1;
//     return ret;
// }

// __global__ void Mat::dot(const Mat& other, Mat* ret){
//     if (this->m_width != other.m_height)
//     {
//         printf("Invalid dot product, shapes do not match {%i, %i} vs {%i, %i}",
//             this->m_height, this->m_width, other.m_height, other.m_width);
//         assert(this->m_width == other.m_height);
//     }
//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;
//     int k = blockDim.z*blockId.z + threadId.z;

//     if (i == 0 && j == 0 && k == 0) {
//         // Initialize the return matrix
//         ret = &Mat(this->m_height, other.m_width);
//     }

//     if (i >= this->m_height || j >= this->m_width || k >= this->m_height) return;
//     ret->m_buffer[i][j] += this->m_buffer[i][k] * other.m_buffer[k][j];
// }

// void Mat::dot(const float* other, int height, Mat* ret)
// {
//     if ((std::size_t)this->m_width != other.size())
//     {
//         printf("Invalid dot product, shapes do not match {%i, %i} vs {%zd, 1}",
//                this->m_height, this->m_width, other.size());
//         throw "Invalid dot product";
//     }
//     Mat vector(other, height);
//     dot(vector, ret); // Fix dot call
// }

// __global__ void Mat::T(Mat* ret) {
//     if (ret == NULL) {
//         // Initialize the return matrix
//         ret = &Mat(this->m_width, this->m_height);
//     }

//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;
//     if (i >= this->m_height || j >= this->m_width) return;
//     ret->m_buffer[j][i] = this->m_buffer[i][j];
// }

// float* Mat::operator[](const int pos) const {
//     return this->m_buffer[pos];
// }

// const float* Mat::operator[](const int pos) {
//     return this->m_buffer[pos];
// }

// __global__ void Mat::operator+(const Mat& other, Mat* ret) const{
//     if ((this->m_width != other.m_width) || (m_height != other.m_height && other.m_height != 1))
//     {
//         printf("Could not add matrices, dimensions do not match {%i, %i} vs {%i, %i}",
//             this->m_height, this->m_width, other.m_height, other.m_width);
//         throw "Invalid addition";
//     }

//     if (ret == NULL) {
//         // Initialize the return matrix
//         ret = Mat(this->m_height, this->m_width);
//     }

//     if (m_height == other.m_height)
//     {
//         int i = blockDim.x*blockId.x + threadId.x;
//         int j = blockDim.y*blockId.y + threadId.y;
//         if (i >= this->m_height || j >= this->m_width) return;    
//         ret->m_buffer[i][j] = this->m_buffer[i][j] + other.m_buffer[i][j];
//     }
//     else
//     {
//         int i = blockDim.x*blockId.x + threadId.x;
//         int j = blockDim.y*blockId.y + threadId.y;
//         if (i >= this->m_height || j >= this->m_width) return;    
//         ret->m_buffer[i][j] = this->m_buffer[i][j] + other.m_buffer[0][j];
//     }
// }

// __global__ void Mat::operator-(const Mat& other, Mat* ret) const{
//     if ((this->m_width != other.m_width) || (m_height != other.m_height && other.m_height != 1))
//     {
//         printf("Could not subtract matrices, dimensions do not match {%i, %i} vs {%i, %i}",
//             this->m_height, this->m_width, other.m_height, other.m_width);
//         throw "Invalid subtraction";
//     }

//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;
    
//     if (i == 0 && j == 0) {
//         // Initialize the return matrix
//         ret = Mat(this->m_height, this->m_width);
//     } 
//     else if (i >= this->m_height || j >= this->m_width) return;
    
//     if (m_height == other.m_height) {
//         ret->m_buffer[i][j] = this->m_buffer[i][j] - other.m_buffer[i][j];
//     }
//     else {
//         ret->m_buffer[i][j] = this->m_buffer[i][j] - other.m_buffer[0][j];
//     }
// }

// __global__ void Mat::operator*(const float& factor, Mat* ret) const{
//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;

//     if (i == 0 && j == 0) {
//         // Initialize the return matrix
//         ret = Mat(this->m_height, this->m_width);
//     } 
//     else if (i >= this->m_height || j >= this->m_width) return;

//     ret.m_buffer[i][j] = factor * this->m_buffer[i][j];
// }

// __global__ void Mat::operator*(const Mat& other, Mat* ret) const{
//     /**
//      * Trying to replicate numpy broadcasting 
//      * https://numpy.org/doc/stable/user/basics.broadcasting.html
//      * TODO
//      */
//     int ret_h = this->m_height;
//     if (other.m_height > ret_h) {
//         ret_h = other.m_height;
//     }
//     int ret_w = this->m_height;
//     if (other.m_height > ret_w) {
//         ret_w = other.m_width;
//     }
    
//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;

//     if (i == 0 && j == 0) {
//         // Initialize the return matrix
//         ret = Mat(ret_h, ret_w);
//     } else if (i >= ret_h || j >= ret_w) return;


//     if (this->m_height == 1 && other.m_width == 1){
//         ret->m_buffer[i][j] += this->m_buffer[0][i] * other.m_buffer[j][0];
//     }
//     else if (this->m_width == 1 && other.m_height == 1){
//         ret.m_buffer[i][j] += other.m_buffer[0][i] * this->m_buffer[j][0];
//     }
//     else{
//         printf("Could not broadcast matrices, dimensions do not match {%i, %i} vs {%i, %i}",
//                this->m_height, this->m_width, other.m_height, other.m_width);
//         assert(0); // Need cleaner asset
//     }
// }

void Mat::print() const {
    std::cout << "{\n";
    for (int i = 0; i < this->m_height; ++i) {
        std::cout << "  { ";
        for (int j = 0; j < this->m_width;) {
            std::cout << this->m_buffer[i * this->m_width + j];
            if (++j < this->m_width)
                std::cout << ", ";
        }
        std::cout << " }\n";
    }
    std::cout << "}\n";
}

// __global__ void Mat::copy(Mat* ret) const {
//     if (ret == NULL) {
//         // Initialize the return matrix
//         ret = Mat(this->m_height, this->m_width);
//     }

//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;
//     if (i >= this->m_height || j >= this->m_width) return;    
//     ret->m_buffer[h][w] = this->m_buffer[h][w];
// }

// float mean_vector(const std::vector<float>& v)
// {
//     float r = 0.;
//     for (std::size_t i = 0; i < v.size(); ++i)
//         r += v[i];
//     return r / v.size();
// }

// Mat Mat::mean() const {
//     std::vector<float> aggreagate(m_width, 0);
//     for (int i = 0; i < m_height; ++i)
//         for (int j = 0; j < m_width; ++j)
//             aggreagate[j] += m_buffer[i][j];
//     for (int i = 0; i < m_width; ++i)
//         aggreagate[i] /= m_height;
//     return Mat(std::vector<std::vector<float>>{aggreagate});
// }

// std::vector<std::tuple<float, Eigen::VectorXf>> get_eigen(Eigen::MatrixXf m) {
//     Eigen::EigenSolver<Eigen::MatrixXf> eigensolver;

//     eigensolver.compute(m);

//     Eigen::VectorXf eigen_values = eigensolver.eigenvalues().real();
//     Eigen::MatrixXf eigen_vectors = eigensolver.eigenvectors().real();
//     std::vector<std::tuple<float, Eigen::VectorXf>> eigen_vectors_and_values;

//     for(int i = 0; i < eigen_values.size(); i++){
//         std::tuple<float, Eigen::VectorXf> vec_and_val(eigen_values[i], eigen_vectors.row(i));
//         eigen_vectors_and_values.push_back(vec_and_val);
//     }

//     std::sort(eigen_vectors_and_values.begin(), eigen_vectors_and_values.end(),
//               [&](const std::tuple<float, Eigen::VectorXf>& a, const std::tuple<float, Eigen::VectorXf>& b) -> bool{
//                   return std::get<0>(a) <= std::get<0>(b);
//               });

//     return eigen_vectors_and_values;
// }

// std::vector<std::tuple<float, std::vector<float>>> Mat::eigen() const {
//     Eigen::MatrixXf eigen_mat(m_height, m_width);

//     for (int i = 0; i < m_height; ++i)
//         for (int j = 0; j < m_width; ++j)
//             eigen_mat(i, j) = m_buffer[i][j];

//     auto eigen_value_vector = get_eigen(eigen_mat);

//     std::vector<std::tuple<float, std::vector<float>>> ret;
//     for (std::size_t i = 0; i < eigen_value_vector.size(); ++i)
//     {
//         std::vector<float> tmp;
//         auto eigen_vector = std::get<1>(eigen_value_vector[i]);
//         for (int j = 0; j < m_height; ++j)
//             tmp.push_back(eigen_vector[j]);
//         std::tuple<float, std::vector<float>> tup = std::make_tuple(std::get<0>(eigen_value_vector[i]), tmp);
//         ret.push_back(tup);
//     }

//     return ret;
// }

// Mat Mat::inverse() const {
//     Eigen::MatrixXf eigen_mat(m_height, m_width);

//     for (int i = 0; i < m_height; ++i)
//         for (int j = 0; j < m_width; ++j)
//             eigen_mat(i, j) = m_buffer[i][j];


//     auto eigen_inverse = eigen_mat.inverse();

//     Mat ret(m_height, m_width);
//     for (int i = 0; i < m_height; ++i) {
//         for (int j = 0; j < m_width; ++j) {
//             ret[i][j] = eigen_inverse(i, j);
//         }
//     }

//     return ret;
// }
