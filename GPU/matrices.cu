#include "hip/hip_runtime.h"
#include "matrices.cuh"
#include <stdlib.h>
#include <assert.h>

#define checkCUDAError(val) { checkError((val), #val, __FILE__, __LINE__); }    // in-line regular function

void checkError(hipError_t code, char const * func, const char *file, const int line)
{
    if (code != hipSuccess) 
    {
        std::cerr << "CUDA error returned from \"" << func << "\" at "
                  << file << ":" << line << "\nError code: " << code
                  << "(" << hipGetErrorString(code) << ")\n";
        hipDeviceReset();
        exit(code);
    }
}

__global__ void mat_init(float* buffer, int height, int width, int value) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i >= width * height) return;

    buffer[i] = value;
}

Mat::Mat(int height, int width)
    : m_height(height)
    , m_width(width)
    , m_buffer((float*) calloc(height * width, sizeof(float)))
{}

// Need to use a custom kernel instead of CudaMemSet because we operate of float pointers
Mat::Mat(int height, int width, float value)
    : m_height{height}
    , m_width{width}
{
    std::size_t buffer_size = height * width;
    this->m_buffer = (float*) malloc(height * width * sizeof(float));
    float* d_buffer;
    checkCUDAError(hipMalloc(&d_buffer, height * width * sizeof(float)));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock) ? buffer_size : prop.maxThreadsPerBlock;
    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;
    mat_init<<<nbBlocks, threadsPerBlock>>>(d_buffer, height, width, value);
    hipDeviceSynchronize();
    checkCUDAError(hipMemcpy(this->m_buffer, d_buffer, height * width * sizeof(float), hipMemcpyDeviceToHost));
    hipFree(d_buffer);
}

Mat::Mat(float* list_init, int height, int width)
    : m_height(height)
    , m_width(width)
{
    std::size_t buffer_size = height * width;
    this->m_buffer = (float*) malloc(buffer_size * sizeof(float));
    checkCUDAError(hipMemcpy(this->m_buffer, list_init, buffer_size * sizeof(float), hipMemcpyHostToHost));
}

Mat::Mat(float* list_init, int width)
    : Mat(list_init, 1, width)
    {}

Mat::Mat(const Mat& m)
    : Mat(m.m_buffer, m.m_height, m.m_width)
    {}

Mat::~Mat(){
    free(this->m_buffer);
}

// I don't think that using a kernel (with all the overhead needed) will be faster than a little for loop
Mat Mat::eye(int dim)
{
    Mat ret(dim, dim);
    for (int i = 0; i < dim; ++i)
        ret.m_buffer[i * ret.m_width + i] = 1;
    return ret;
}

// Internet say, use a loop for k to avoid concurrency problem
__global__ void dot_kernel(float* self, float* other, float* ret,
                           int s_height, int s_width, int o_width){
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * o_width) return;

    int i = th / o_width; //0 to height
    int j = th % o_width; //0 to width
    for (int k = 0; k < s_width; ++k)
        ret[i * o_width + j] += self[i * s_width + k] * other[k * o_width + j];
}

Mat Mat::dot(const Mat& other)
{
    if (m_width != other.m_height)
    {
        printf("Invalid dot product, shapes do not match {%i, %i} vs {%i, %i}",
               m_height, m_width, other.m_height, other.m_width);
        throw "Invalid dot product";
    }

    Mat ret(m_height, other.m_width);
    float* ret_buffer;
    checkCUDAError(hipMalloc(&ret_buffer, ret.m_height * ret.m_width* sizeof(float)));

    float* self_buffer;
    checkCUDAError(hipMalloc(&self_buffer, m_height * m_width* sizeof(float)));
    checkCUDAError(hipMemcpy(self_buffer, m_buffer, m_height * m_width * sizeof(float), hipMemcpyHostToDevice));
    float* other_buffer;
    checkCUDAError(hipMalloc(&other_buffer, other.m_height * other.m_width * sizeof(float)));
    checkCUDAError(hipMemcpy(other_buffer, other.m_buffer,
                              other.m_height * other.m_width * sizeof(float), hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t buffer_size = ret.m_height * ret.m_width;
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock)
        ? buffer_size : prop.maxThreadsPerBlock;
    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;
    dot_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, other_buffer, ret_buffer,
                                              m_height, m_width, other.m_width);
    hipDeviceSynchronize();

    checkCUDAError(hipMemcpy(ret.m_buffer, ret_buffer, ret.m_height * ret.m_width * sizeof(float),
                              hipMemcpyDeviceToHost));

    hipFree(ret_buffer);
    hipFree(self_buffer);
    hipFree(other_buffer);
    return ret;
}

__global__ void T_kernel(float* self, float* ret, int s_height, int s_width) {
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * s_width) return;

    int i = th / s_width; //0 to height
    int j = th % s_width; //0 to width
    ret[j * s_height + i] = self[i * s_width + j];
}

Mat Mat::T() {
    Mat ret(m_width, m_height);
    float* ret_buffer;
    checkCUDAError(hipMalloc(&ret_buffer, ret.m_height * ret.m_width * sizeof(float)));

    float* self_buffer;
    checkCUDAError(hipMalloc(&self_buffer, m_height * m_width* sizeof(float)));
    checkCUDAError(hipMemcpy(self_buffer, m_buffer, m_height * m_width * sizeof(float), hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t buffer_size = ret.m_height * ret.m_width;
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock)
        ? buffer_size : prop.maxThreadsPerBlock;
    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;
    T_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, ret_buffer, m_height, m_width);
    hipDeviceSynchronize();

    checkCUDAError(hipMemcpy(ret.m_buffer, ret_buffer, ret.m_height * ret.m_width * sizeof(float),
                              hipMemcpyDeviceToHost));

    hipFree(ret_buffer);
    hipFree(self_buffer);
    return ret;
}

__global__ void add_kernel(float* self, float* other, float* ret, int s_height, int s_width) {
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * s_width) return;
    ret[th] = self[th] + other[th];
}

__global__ void add_broadcast_kernel(float* self, float* other, float* ret, int s_height, int s_width) {
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * s_width) return;
    int i = th / s_width; //0 to height
    int j = th % s_width; //0 to width
    ret[i * s_width + j] = self[i * s_width + j] + other[j];
}

Mat Mat::operator+(const Mat& other) const{
    if ((this->m_width != other.m_width) || (m_height != other.m_height && other.m_height != 1))
    {
        printf("Could not add matrices, dimensions do not match {%i, %i} vs {%i, %i}",
            this->m_height, this->m_width, other.m_height, other.m_width);
        throw "Invalid addition";
    }


    Mat ret(m_height, m_width);
    float* ret_buffer;
    checkCUDAError(hipMalloc(&ret_buffer, ret.m_height * ret.m_width* sizeof(float)));

    float* self_buffer;
    checkCUDAError(hipMalloc(&self_buffer, m_height * m_width* sizeof(float)));
    checkCUDAError(hipMemcpy(self_buffer, m_buffer, m_height * m_width * sizeof(float), hipMemcpyHostToDevice));
    float* other_buffer;
    checkCUDAError(hipMalloc(&other_buffer, other.m_height * other.m_width * sizeof(float)));
    checkCUDAError(hipMemcpy(other_buffer, other.m_buffer,
                              other.m_height * other.m_width * sizeof(float), hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t buffer_size = ret.m_height * ret.m_width;
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock)
        ? buffer_size : prop.maxThreadsPerBlock;
    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;

    if (m_height == other.m_height)
        add_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, other_buffer, ret_buffer,
                                                  m_height, m_width);
    else
        add_broadcast_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, other_buffer, ret_buffer,
                                                  m_height, m_width);

    hipDeviceSynchronize();

    checkCUDAError(hipMemcpy(ret.m_buffer, ret_buffer, ret.m_height * ret.m_width * sizeof(float),
                              hipMemcpyDeviceToHost));

    hipFree(ret_buffer);
    hipFree(self_buffer);
    hipFree(other_buffer);
    return ret;
}

__global__ void sub_kernel(float* self, float* other, float* ret, int s_height, int s_width) {
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * s_width) return;
    ret[th] = self[th] - other[th];
}

__global__ void sub_broadcast_kernel(float* self, float* other, float* ret, int s_height, int s_width) {
    int th = blockDim.x * blockIdx.x + threadIdx.x;

    if (th >= s_height * s_width) return;
    int i = th / s_width; //0 to height
    int j = th % s_width; //0 to width
    ret[i * s_width + j] = self[i * s_width + j] - other[j];
}

Mat Mat::operator-(const Mat& other) const{
    if ((this->m_width != other.m_width) || (m_height != other.m_height && other.m_height != 1))
    {
        printf("Could not add matrices, dimensions do not match {%i, %i} vs {%i, %i}",
            this->m_height, this->m_width, other.m_height, other.m_width);
        throw "Invalid addition";
    }


    Mat ret(m_height, m_width);
    float* ret_buffer;
    checkCUDAError(hipMalloc(&ret_buffer, ret.m_height * ret.m_width* sizeof(float)));

    float* self_buffer;
    checkCUDAError(hipMalloc(&self_buffer, m_height * m_width* sizeof(float)));
    checkCUDAError(hipMemcpy(self_buffer, m_buffer, m_height * m_width * sizeof(float), hipMemcpyHostToDevice));
    float* other_buffer;
    checkCUDAError(hipMalloc(&other_buffer, other.m_height * other.m_width * sizeof(float)));
    checkCUDAError(hipMemcpy(other_buffer, other.m_buffer,
                              other.m_height * other.m_width * sizeof(float), hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::size_t buffer_size = ret.m_height * ret.m_width;
    std::size_t threadsPerBlock = (buffer_size < prop.maxThreadsPerBlock)
        ? buffer_size : prop.maxThreadsPerBlock;
    std::size_t nbBlocks = buffer_size / threadsPerBlock + 1;

    if (m_height == other.m_height)
        sub_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, other_buffer, ret_buffer,
                                                  m_height, m_width);
    else
        sub_broadcast_kernel<<<nbBlocks, threadsPerBlock>>>(self_buffer, other_buffer, ret_buffer,
                                                  m_height, m_width);

    hipDeviceSynchronize();

    checkCUDAError(hipMemcpy(ret.m_buffer, ret_buffer, ret.m_height * ret.m_width * sizeof(float),
                              hipMemcpyDeviceToHost));

    hipFree(ret_buffer);
    hipFree(self_buffer);
    hipFree(other_buffer);
    return ret;
}

// __global__ void Mat::operator-(const Mat& other, Mat* ret) const{
//     if ((this->m_width != other.m_width) || (m_height != other.m_height && other.m_height != 1))
//     {
//         printf("Could not subtract matrices, dimensions do not match {%i, %i} vs {%i, %i}",
//             this->m_height, this->m_width, other.m_height, other.m_width);
//         throw "Invalid subtraction";
//     }

//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;
    
//     if (i == 0 && j == 0) {
//         // Initialize the return matrix
//         ret = Mat(this->m_height, this->m_width);
//     } 
//     else if (i >= this->m_height || j >= this->m_width) return;
    
//     if (m_height == other.m_height) {
//         ret->m_buffer[i][j] = this->m_buffer[i][j] - other.m_buffer[i][j];
//     }
//     else {
//         ret->m_buffer[i][j] = this->m_buffer[i][j] - other.m_buffer[0][j];
//     }
// }

// __global__ void Mat::operator*(const float& factor, Mat* ret) const{
//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;

//     if (i == 0 && j == 0) {
//         // Initialize the return matrix
//         ret = Mat(this->m_height, this->m_width);
//     } 
//     else if (i >= this->m_height || j >= this->m_width) return;

//     ret.m_buffer[i][j] = factor * this->m_buffer[i][j];
// }

// __global__ void Mat::operator*(const Mat& other, Mat* ret) const{
//     /**
//      * Trying to replicate numpy broadcasting 
//      * https://numpy.org/doc/stable/user/basics.broadcasting.html
//      * TODO
//      */
//     int ret_h = this->m_height;
//     if (other.m_height > ret_h) {
//         ret_h = other.m_height;
//     }
//     int ret_w = this->m_height;
//     if (other.m_height > ret_w) {
//         ret_w = other.m_width;
//     }
    
//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;

//     if (i == 0 && j == 0) {
//         // Initialize the return matrix
//         ret = Mat(ret_h, ret_w);
//     } else if (i >= ret_h || j >= ret_w) return;


//     if (this->m_height == 1 && other.m_width == 1){
//         ret->m_buffer[i][j] += this->m_buffer[0][i] * other.m_buffer[j][0];
//     }
//     else if (this->m_width == 1 && other.m_height == 1){
//         ret.m_buffer[i][j] += other.m_buffer[0][i] * this->m_buffer[j][0];
//     }
//     else{
//         printf("Could not broadcast matrices, dimensions do not match {%i, %i} vs {%i, %i}",
//                this->m_height, this->m_width, other.m_height, other.m_width);
//         assert(0); // Need cleaner asset
//     }
// }

void Mat::print() const {
    std::cout << "{\n";
    for (int i = 0; i < this->m_height; ++i) {
        std::cout << "  { ";
        for (int j = 0; j < this->m_width;) {
            std::cout << this->m_buffer[i * this->m_width + j];
            if (++j < this->m_width)
                std::cout << ", ";
        }
        std::cout << " }\n";
    }
    std::cout << "}\n";
}

// __global__ void Mat::copy(Mat* ret) const {
//     if (ret == NULL) {
//         // Initialize the return matrix
//         ret = Mat(this->m_height, this->m_width);
//     }

//     int i = blockDim.x*blockId.x + threadId.x;
//     int j = blockDim.y*blockId.y + threadId.y;
//     if (i >= this->m_height || j >= this->m_width) return;    
//     ret->m_buffer[h][w] = this->m_buffer[h][w];
// }

// float mean_vector(const std::vector<float>& v)
// {
//     float r = 0.;
//     for (std::size_t i = 0; i < v.size(); ++i)
//         r += v[i];
//     return r / v.size();
// }

// Mat Mat::mean() const {
//     std::vector<float> aggreagate(m_width, 0);
//     for (int i = 0; i < m_height; ++i)
//         for (int j = 0; j < m_width; ++j)
//             aggreagate[j] += m_buffer[i][j];
//     for (int i = 0; i < m_width; ++i)
//         aggreagate[i] /= m_height;
//     return Mat(std::vector<std::vector<float>>{aggreagate});
// }

// std::vector<std::tuple<float, Eigen::VectorXf>> get_eigen(Eigen::MatrixXf m) {
//     Eigen::EigenSolver<Eigen::MatrixXf> eigensolver;

//     eigensolver.compute(m);

//     Eigen::VectorXf eigen_values = eigensolver.eigenvalues().real();
//     Eigen::MatrixXf eigen_vectors = eigensolver.eigenvectors().real();
//     std::vector<std::tuple<float, Eigen::VectorXf>> eigen_vectors_and_values;

//     for(int i = 0; i < eigen_values.size(); i++){
//         std::tuple<float, Eigen::VectorXf> vec_and_val(eigen_values[i], eigen_vectors.row(i));
//         eigen_vectors_and_values.push_back(vec_and_val);
//     }

//     std::sort(eigen_vectors_and_values.begin(), eigen_vectors_and_values.end(),
//               [&](const std::tuple<float, Eigen::VectorXf>& a, const std::tuple<float, Eigen::VectorXf>& b) -> bool{
//                   return std::get<0>(a) <= std::get<0>(b);
//               });

//     return eigen_vectors_and_values;
// }

// std::vector<std::tuple<float, std::vector<float>>> Mat::eigen() const {
//     Eigen::MatrixXf eigen_mat(m_height, m_width);

//     for (int i = 0; i < m_height; ++i)
//         for (int j = 0; j < m_width; ++j)
//             eigen_mat(i, j) = m_buffer[i][j];

//     auto eigen_value_vector = get_eigen(eigen_mat);

//     std::vector<std::tuple<float, std::vector<float>>> ret;
//     for (std::size_t i = 0; i < eigen_value_vector.size(); ++i)
//     {
//         std::vector<float> tmp;
//         auto eigen_vector = std::get<1>(eigen_value_vector[i]);
//         for (int j = 0; j < m_height; ++j)
//             tmp.push_back(eigen_vector[j]);
//         std::tuple<float, std::vector<float>> tup = std::make_tuple(std::get<0>(eigen_value_vector[i]), tmp);
//         ret.push_back(tup);
//     }

//     return ret;
// }

// Mat Mat::inverse() const {
//     Eigen::MatrixXf eigen_mat(m_height, m_width);

//     for (int i = 0; i < m_height; ++i)
//         for (int j = 0; j < m_width; ++j)
//             eigen_mat(i, j) = m_buffer[i][j];


//     auto eigen_inverse = eigen_mat.inverse();

//     Mat ret(m_height, m_width);
//     for (int i = 0; i < m_height; ++i) {
//         for (int j = 0; j < m_width; ++j) {
//             ret[i][j] = eigen_inverse(i, j);
//         }
//     }

//     return ret;
// }
