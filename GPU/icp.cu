#include "hip/hip_runtime.h"
#include "icp.cuh"

icp::icp(const Mat& src, const Mat& ref)
    : src_(src)
    , ref_(ref)
    , src_transformed_(src.copy())
    , translation_scalars_(Mat(1, 3))
{
    this->rotation_matrix_ = (Mat**)calloc(3, sizeof(Mat*));
    for (int i = 0; i < 3; i++)
        this->rotation_matrix_[i] = new Mat(3, 3);
}

icp::~icp() {
    for (int i = 0; i < 3; i++)
        delete rotation_matrix_[i];
    free(this->rotation_matrix_);
}

void icp::set_rotation_matrix(const float theta1, const float theta2, const float theta3) const {
    rotation_matrix_[0]->m_buffer[0] = 1;
    rotation_matrix_[0]->m_buffer[4] = std::cos(theta1);
    rotation_matrix_[0]->m_buffer[5] = -std::sin(theta1);
    rotation_matrix_[0]->m_buffer[7] = std::sin(theta1);
    rotation_matrix_[0]->m_buffer[8] = std::cos(theta1);

    rotation_matrix_[1]->m_buffer[0] = std::cos(theta2);
    rotation_matrix_[1]->m_buffer[2] = std::sin(theta2);
    rotation_matrix_[1]->m_buffer[4] = 1;
    rotation_matrix_[1]->m_buffer[6] = -std::sin(theta2);
    rotation_matrix_[1]->m_buffer[8] = std::cos(theta2);

    rotation_matrix_[2]->m_buffer[0] = std::cos(theta3);
    rotation_matrix_[2]->m_buffer[1] = -std::sin(theta3);
    rotation_matrix_[2]->m_buffer[3] = std::sin(theta3);
    rotation_matrix_[2]->m_buffer[4] = std::cos(theta3);
    rotation_matrix_[2]->m_buffer[8] = 1;
}

Mat** icp::get_dr(const float theta1, const float theta2, const float theta3) const {
    Mat** res = (Mat**)calloc(3, sizeof(Mat*));
    for (int i = 0; i < 3; i++)
        res[i] = new Mat(3, 3);

    res[0]->m_buffer[4] = -std::sin(theta1);
    res[0]->m_buffer[5] = -std::cos(theta1);
    res[0]->m_buffer[7] = std::cos(theta1);
    res[0]->m_buffer[8] = -std::sin(theta1);

    res[1]->m_buffer[0] = -std::sin(theta2);
    res[1]->m_buffer[2] = std::cos(theta2);
    res[1]->m_buffer[6] = -std::cos(theta2);
    res[1]->m_buffer[8] = -std::sin(theta2);

    res[2]->m_buffer[0] = -std::sin(theta3);
    res[2]->m_buffer[1] = -std::cos(theta3);
    res[2]->m_buffer[3] = std::cos(theta3);
    res[2]->m_buffer[4] = -std::sin(theta3);

    return res;
}

Mat icp::get_jacobian(const Mat& x, const Mat& p_point) const {
    Mat jacobian = Mat(3, 6);
    jacobian.m_buffer[0] = 1;
    jacobian.m_buffer[7] = 1;
    jacobian.m_buffer[14] = 1;

    set_rotation_matrix(x.m_buffer[3], x.m_buffer[4], x.m_buffer[5]);
    auto dr = this->get_dr(x.m_buffer[3], x.m_buffer[4], x.m_buffer[5]);

    auto jacob1 = dr[0]->dot(*rotation_matrix_[1]).dot(*rotation_matrix_[2]).dot(p_point);
    auto jacob2 = rotation_matrix_[0]->dot(*dr[1]).dot(*rotation_matrix_[2]).dot(p_point);
    auto jacob3 = rotation_matrix_[0]->dot(*rotation_matrix_[1]).dot(*dr[2]).dot(p_point);

    for (int i = 0; i < 3; i++) {
        jacobian.m_buffer[i * jacobian.m_width + 3] = jacob1.m_buffer[i * jacob1.m_width + 0];
        jacobian.m_buffer[i * jacobian.m_width + 4] = jacob2.m_buffer[i * jacob2.m_width + 0];
        jacobian.m_buffer[i * jacobian.m_width + 5] = jacob3.m_buffer[i * jacob3.m_width + 0];
    }


    for (int i = 0; i < 3; i++)
        delete dr[i];
    free(dr);
    return jacobian;
}

Mat icp::err(const Mat& x, const Mat& p_point, const Mat& q_point) const {
    set_rotation_matrix(x.m_buffer[3], x.m_buffer[4], x.m_buffer[5]);
    auto translation = Mat(3, 1);
    translation.m_buffer[0] = x.m_buffer[0];
    translation.m_buffer[1] = x.m_buffer[1];
    translation.m_buffer[2] = x.m_buffer[2];
    auto prediction = rotation_matrix_[0]->dot(*rotation_matrix_[1])
        .dot(*rotation_matrix_[2]).dot(p_point) + translation;
    return prediction - q_point;
}

prep_sys_t* icp::prepare_system(Mat& x, Mat& P, Mat& Q) const {
    Mat H = Mat(6,6);
    Mat G = Mat(6,1);
    float chi = 0.;
    for (int i = 0; i < P.m_height; ++i) {
        auto p_width = P.m_width;
        auto q_width = Q.m_width;

        Mat p_point(p_width, 1);
        Mat q_point(q_width, 1);
        for (int j = 0; j < p_width; j++) {
            p_point.m_buffer[j] = P.m_buffer[i * p_width + j];
        }
        for (int j = 0; j < q_width; j++) {
            q_point.m_buffer[j] = Q.m_buffer[i * q_width + j];
        }

        auto e = this->err(x, p_point, q_point);
        auto J = this->get_jacobian(x, p_point);
        H = H + J.T().dot(J);
        G = G + J.T().dot(e);
        chi += e.T().dot(e).m_buffer[0];
    }
    prep_sys_t* res = (prep_sys_t*)malloc(sizeof(prep_sys_t));
    res->h = new Mat(H);
    res->g = new Mat(G);
    res->chi = chi;
    return res;
}

// Compute the 3 rotation matrix and the 3 translation scalars to transform src_ in ref_
icp& icp::fit(int iterations, float threshold, bool force_iteration){
    auto x = Mat(1,6); // 3 rotation factors + 3 translation
    float chi = 0.;
    int i = 0;
    for (; i < iterations; ++i){
        set_rotation_matrix(x.m_buffer[3], x.m_buffer[4], x.m_buffer[5]);
        auto prep_sys = prepare_system(x, src_, ref_);
        Mat H = *prep_sys->h;
        Mat G = *prep_sys->g;
        float chi = prep_sys->chi;
        auto dx = H.inverse().dot(G).T();
        x = x - dx;
        translation_scalars_ = Mat(1, 3);
        translation_scalars_.m_buffer[0] = x.m_buffer[0];
        translation_scalars_.m_buffer[1] = x.m_buffer[1];
        translation_scalars_.m_buffer[2] = x.m_buffer[2];
        src_transformed_ = rotation_matrix_[0]->dot(*rotation_matrix_[1])
            .dot(*rotation_matrix_[2]).dot(src_.T()).T() + translation_scalars_;
        delete prep_sys->h;
        delete prep_sys->g;
        free(prep_sys);
        if (chi < threshold && not force_iteration)
            break;
    }
    if (chi >= threshold)
        std::cerr << "ICP did not converge in " << iterations << " iterations, and have a chi value of " << chi << "\n";
    else
        std::cerr << "ICP converge in " << i << " iterations, and have a chi value of " << chi << "\n";

    return *this;
}
