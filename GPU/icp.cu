#include "hip/hip_runtime.h"
#include "icp.hh"

icp::icp(const Mat& src, const Mat& ref)
    : src_(src)
    , ref_(ref)
    , src_transformed_(src.copy())
    , translation_scalars_(Mat(1, 3)) 
{
    this->rotation_matrix_ = (Mat*)calloc(3, sizeof(Mat));
    for (int i = 0; i < 3; i++)
        this->rotation_matrix_[i] = Mat(3, 3);
}

icp::~icp() {
    free(this->rotation_matrix_);
}

Mat* icp::get_r(const float theta1, const float theta2, const float theta3) const {
    Mat* res = (Mat*)calloc(3, sizeof(Mat));
    
    float *vx = (float*)calloc(9, sizeof(float));
    vx[0] = 1;
    vx[4] = std::cos(theta1);
    vx[5] = -std::sin(theta1);
    vx[7] = std::sin(theta1);
    vx[8] = std::cos(theta1);
    res[0] = Mat(vx, 3, 3);

    float *vy = (float*)calloc(9, sizeof(float));
    vy[0] = std::cos(theta2);
    vy[2] = std::sin(theta2);
    vy[4] = 1;
    vy[6] = -std::sin(theta2);
    vy[8] = std::cos(theta2);
    res[1] = Mat(vy, 3, 3);

    float *vz = (float*)calloc(9, sizeof(float));
    vz[0] = std::cos(theta3);
    vz[1] = -std::sin(theta3);
    vz[3] = std::sin(theta3);
    vz[4] = std::cos(theta3);
    vz[8] = 1;
    res[2] = Mat(vz, 3, 3);

    return res;
}

Mat* icp::get_dr(const float theta1, const float theta2, const float theta3) const {
    Mat* res = (Mat*)calloc(3, sizeof(Mat));
    
    float *vx = (float*)calloc(9, sizeof(float));
    vx[4] = -std::sin(theta1);
    vx[5] = -std::cos(theta1);
    vx[7] = std::cos(theta1);
    vx[8] = -std::sin(theta1);
    res[0] = Mat(vx, 3, 3);

    float *vy = (float*)calloc(9, sizeof(float));
    vy[0] = -std::sin(theta2);
    vy[2] = std::cos(theta2);
    vy[6] = -std::cos(theta2);
    vy[8] = -std::sin(theta2);
    res[1] = Mat(vy, 3, 3);

    float *vz = (float*)calloc(9, sizeof(float));
    vz[0] = -std::sin(theta3);
    vz[1] = -std::cos(theta3);
    vz[3] = std::cos(theta3);
    vz[4] = -std::sin(theta3);
    res[2] = Mat(vz, 3, 3);

    return res;
}

Mat icp::get_jacobian(const Mat& x, const Mat& p_point) const {
    float* v = (float*)calloc(3*6, sizeof(float));
    v[0] = 1;
    v[7] = 1;
    v[14] = 1;
    Mat jacobian = Mat(v, 3, 6);
    
    auto r = this->get_r(x.m_buffer[3], x.m_buffer[4], x.m_buffer[5]);
    auto dr = this->get_dr(x.m_buffer[3], x.m_buffer[4], x.m_buffer[5]);

    auto jacob1 = dr[0].dot(r[1]).dot(r[2]).dot(p_point);
    auto jacob2 = r[0].dot(dr[1]).dot(r[2]).dot(p_point);
    auto jacob3 = r[0].dot(r[1]).dot(dr[2]).dot(p_point);

    for (int i = 0; i < 3; i++) {
        jacobian.m_buffer[i * jacobian.m_width + 3] = jacob1.m_buffer[i * jacob1.m_width + 0];
        jacobian.m_buffer[i * jacobian.m_width + 4] = jacob2.m_buffer[i * jacob2.m_width + 0];
        jacobian.m_buffer[i * jacobian.m_width + 5] = jacob3.m_buffer[i * jacob3.m_width + 0];
    }

    return jacobian;
}



// float norm(const Mat& p, int size) {
//     float r = 0;
//     for (int i = 0; i < size; i++) {
//         r += p.m_buffer[i] * p.m_buffer[i];
//     }
//     return std::sqrt(r);
// }

Mat icp::err(const Mat& x, const Mat& p_point, const Mat& q_point) const {
    auto rotation = this->get_r(x.m_buffer[3], x.m_buffer[4], x.m_buffer[5]);
    float* vt = (float*)calloc(3, sizeof(float));
    vt[0] = x.m_buffer[0];
    vt[1] = x.m_buffer[1];
    vt[2] = x.m_buffer[2];
    auto translation = Mat(vt, 3);
    auto prediction = rotation[0].dot(rotation[1]).dot(rotation[2]).dot(p_point) + translation;
    return prediction - q_point;
}

prep_sys_t icp::prepare_system(Mat& x, Mat& P, Mat& Q) const {
    Mat H = Mat(6,6);
    Mat G = Mat(6,1);
    float chi = 0.;
    for (int i = 0; i < P.m_height; ++i) {
        auto p_width = P.m_width;
        auto q_width = Q.m_width;
        float* vp = (float*)calloc(p_width, sizeof(float));
        float* vq = (float*)calloc(q_width, sizeof(float));
        for (int j = 0; j < p_width; j++) {
            vp[j] = P.m_buffer[i * p_width + j];
        }
        for (int j = 0; j < q_width; j++) {
            vq[j] = Q.m_buffer[i * q_width + j];
        }
        Mat p_point = Mat(vp, p_width);
        Mat q_point = Mat(vq, q_width);
        auto e = this->err(x, p_point, q_point);
        auto J = this->get_jacobian(x, p_point);
        H = H + J.T().dot(J);
        G = G + J.T().dot(e);
        chi += e.T().dot(e).m_buffer[0];
    }
    prep_sys_t* res = (prep_sys_t*)calloc(1, sizeof(prep_sys_t));
    res->h = H;
    res->g = G;
    res->chi = chi;
    return res;
}

// Compute the 3 rotation matrix and the 3 translation scalars to transform src_ in ref_
icp& icp::fit(int iterations, float threshold){
    auto x = Mat(1,6); // 3 rotation factors + 3 translation
    float chi = 0.;
    int i = 0;
    for (; i < iterations; ++i){
        rotation_matrix_ = get_r(x.m_buffer[3], x.m_buffer[4], x.m_buffer[5]);
        auto prep_sys = prepare_system(x, src_, ref_);
        auto H = prep_sys.h;
        auto G = prep_sys.g;
        chi = prep_sys.chi;
        auto dx = H.inverse().dot(G).T();
        x = x - dx;
        float* v = (float*)calloc(3, sizeof(float));
        v[0] = x.m_buffer[0];
        v[1] = x.m_buffer[1];
        v[2] = x.m_buffer[2];
        translation_scalars_ = Mat(v, 3).T();
        auto r_width = rotation_matrix_.m_width;
        float* vr0 = (float*)calloc(r_width, sizeof(float));
        float* vr1 = (float*)calloc(r_width, sizeof(float));
        float* vr2 = (float*)calloc(r_width, sizeof(float));

        for (int i = 0; i < r_width; i++) {
            vr0[i] = rotation_matrix_[i];
            vr1[i] = rotation_matrix_[r_width + i];
            vr2[i] = rotation_matrix_[2 * r_width + i];
        }
        Mat r0 = Mat(vr0, r_width);
        Mat r1 = Mat(vr1, r_width);
        Mat r2 = Mat(vr2, r_width);
        src_transformed_ = r0.dot(r1).dot(r2).dot(src_.T()).T() + translation_scalars_;
        if (chi < threshold)
            break;
    }
    if (chi >= threshold)
        std::cerr << "ICP did not converge in " << iterations << " iterations, and have a chi value of " << chi << "\n";
    else
        std::cerr << "ICP converge in " << i << " iterations, and have a chi value of " << chi << "\n";

    return *this;
}
